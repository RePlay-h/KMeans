
#include "hip/hip_runtime.h"

#include <iostream>
#include <cassert>

#define CUDA_CHECK(ans) {cudaAssert(ans, __FILE__, __LINE__);}

inline void cudaAssert(hipError_t ans, const char *file, unsigned long line, bool is_abort=true) {
    
    if(ans != hipSuccess) {

        std::cerr << "GPU error: " << hipGetErrorString(ans) << ' ' << file << ' ' << line << '\n';

        assert(is_abort);

    }

}

